#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <iterator>
#include <vector>
#include <chrono>

__global__ void Kernel( int SizeX , int SizeY , const float2 * SiteArray , const int * Ping , int * Pong , int k , int * Mutex )
{
    //
    const int CellX = threadIdx.x + blockIdx.x * blockDim.x ;
    const int CellY = threadIdx.y + blockIdx.y * blockDim.y ;

    const int CellIdx = CellX + CellY * SizeX ;
    const int Seed = Ping[CellIdx] ;
    if ( Seed < 0 )
    {
        return ;
    }

    //
    const int2 OffsetArray[8] = { { - 1 , - 1 } ,
                                  {   0 , - 1 } ,
                                  {   1 , - 1 } ,
                                  { - 1 ,   0 } ,
                                  {   1 ,   0 } ,
                                  { - 1 ,   1 } ,
                                  {   0 ,   1 } ,
                                  {   1 ,   1 } } ;

    for ( int i = 0 ; i < 8 ; ++ i )
    {
        const int FillCellX = CellX + k * OffsetArray[i].x ;
        const int FillCellY = CellY + k * OffsetArray[i].y ; 
        if ( FillCellX >= 0 && FillCellX < SizeX && FillCellY >= 0 && FillCellY < SizeY )
        {
            //
            const int FillCellIdx = FillCellX + FillCellY * SizeX ;

            // Lock
            //
            while ( atomicCAS( Mutex , - 1 , FillCellIdx ) == FillCellIdx )
            {
            }

            const int FillSeed = Pong[FillCellIdx] ;

            if ( FillSeed < 0 )
            {
                Pong[FillCellIdx] = Seed ;
            }
            else
            {
                float2 P = make_float2( FillCellX + 0.5f , FillCellY + 0.5f ) ;

                float2 A = SiteArray[Seed] ;
                float2 PA = make_float2( A.x - P.x , A.y - P.y ) ;
                float PALength = PA.x * PA.x + PA.y * PA.y ;

                const float2 B = SiteArray[FillSeed] ;
                float2 PB = make_float2( B.x - P.x , B.y - P.y ) ;
                float PBLength = PB.x * PB.x + PB.y * PB.y ;

                if ( PALength < PBLength )
                {
                    Pong[FillCellIdx] = Seed ;
                }
            }

            // Release
            //
            atomicExch( Mutex , - 1 ) ;
        }
    }
}

int main( int Argc , char * Argv[] )
{
    -- Argc , ++ Argv ;
    if ( Argc != 3 )
    {
        printf("SOMETHING IS WRONG") ;
        return EXIT_FAILURE ;
    }

    //numSeeds - Number of Seeds
    //Size - Voronoi grid size
    int numSeeds = atoi( Argv[0] ) ;
    int Size     = atoi( Argv[1] ) ;

    //
    int NumCudaDevice = 0 ;
    hipGetDeviceCount( & NumCudaDevice ) ;
    if ( ! NumCudaDevice )
    {
        return EXIT_FAILURE ;
    }

    //1. Generate x and y position values for the seeds in seedVec
    //2. Randomly assign seed number to some grid points(x.y) in the voronoiVec
    //3. Assign randomly generated colours to each of the seeds in randomcolourVec
    std::vector< float2 > seedVec ;
    std::vector< int >    voronoiVec( Size * Size , - 1 ) ;
    std::vector< uchar3 > randomColorVec ;
    for ( int i = 0 ; i < numSeeds ; ++ i )
    {
        float X = static_cast< float >( rand() ) / RAND_MAX * Size ;
        float Y = static_cast< float >( rand() ) / RAND_MAX * Size ;
        int CellX = static_cast< int >( floorf( X ) ) ;
        int CellY = static_cast< int >( floorf( Y ) ) ;

        seedVec.push_back( make_float2( CellX + 0.5f , CellY + 0.5f ) ) ;
        voronoiVec[CellX + CellY * Size] = i ;
        //printf("SOMETHING IS GOOD");

        randomColorVec.push_back( make_uchar3( static_cast< unsigned char >( static_cast< float >( rand() ) / RAND_MAX * 255.0f ) ,
                                               static_cast< unsigned char >( static_cast< float >( rand() ) / RAND_MAX * 255.0f ) ,
                                               static_cast< unsigned char >( static_cast< float >( rand() ) / RAND_MAX * 255.0f ) ) ) ;
    }

    //
    size_t seedSize = numSeeds * sizeof( float2 ) ;

    float2 * seedArray = NULL ;
    hipMalloc( & seedArray , seedSize ) ;
    hipMemcpy( seedArray , & seedVec[0] , seedSize , hipMemcpyHostToDevice ) ;

    //BufferSize - Voronoi grid size (Size * Size)
    size_t BufferSize = Size * Size * sizeof( int ) ;

    int * Ping = NULL , * Pong = NULL ;
    hipMalloc( & Ping , BufferSize ) , hipMemcpy( Ping , & voronoiVec[0] , BufferSize , hipMemcpyHostToDevice ) ;
    hipMalloc( & Pong , BufferSize ) , hipMemcpy( Pong , Ping , BufferSize , hipMemcpyDeviceToDevice ) ;

    //Mutex will be used in the kernel to lock and unlock atomic operation
    int * Mutex = NULL ;
    hipMalloc( & Mutex , sizeof( int ) ) , hipMemset( Mutex , - 1 , sizeof( int ) ) ;

    //
    //
    hipDeviceProp_t CudaDeviceProperty ;
    hipGetDeviceProperties( & CudaDeviceProperty , 0 ) ;

    //warpsize = 32 threads
    dim3 BlockDim( CudaDeviceProperty.warpSize , CudaDeviceProperty.warpSize ) ; 
    dim3 GridDim( ( Size + BlockDim.x - 1 ) / BlockDim.x ,
                  ( Size + BlockDim.y - 1 ) / BlockDim.y ) ;

    //run JFA for logn rounds 
    auto start = std::chrono::high_resolution_clock::now();
    for ( int k = Size / 2 ; k > 0 ; k = k >> 1 )
    {
        Kernel<<< GridDim , BlockDim >>>( Size , Size , seedArray , Ping , Pong , k , Mutex ) ;
        hipDeviceSynchronize() ;

        hipMemcpy( Ping , Pong , BufferSize , hipMemcpyDeviceToDevice ) ;
        std::swap( Ping , Pong ) ;
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop -start);
    printf("Execution time %ld microseconds\n", duration.count());

    hipMemcpy( & voronoiVec[0] , Pong , BufferSize , hipMemcpyDeviceToHost ) ;

    //
    hipFree( seedArray ) ;
    hipFree( Ping ) ;
    hipFree( Pong ) ;
    hipFree( Mutex ) ;

    //
    //
    FILE * Output = fopen( Argv[2], "wb" ) ;
    fprintf( Output , "P6\n%d %d\n255\n" , Size , Size ) ;

    std::vector< uchar3 > Pixels( Size * Size ) ;
    for ( int y = 0 ; y < Size ; ++ y )
    {
        for ( int x = 0 ; x < Size ; ++ x )
        {
            const int Seed = voronoiVec[x + y * Size] ;
            if ( Seed != - 1 )
            {
                Pixels[x + y * Size] = randomColorVec[Seed] ;
            }
        }
    }

    for( std::vector< float2 >::const_iterator itr = seedVec.begin() ; itr != seedVec.end() ; ++ itr )
    {
        const int x = static_cast< int >( floorf( itr->x ) ) ;
        const int y = static_cast< int >( floorf( itr->y ) ) ;
        Pixels[x + y * Size] = make_uchar3( 255 , 0 , 0 ) ;
    }

    fwrite( & Pixels[0].x , 3 , Pixels.size() , Output ) ;
    fclose( Output ) ;

    return EXIT_SUCCESS ;
}